#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <chrono>
#include <cstdlib>
#include <algorithm>
#include <cmath>
#include <random>
#include <thrust/device_vector.h> // adicionado
#include <thrust/transform.h> // adicionado
#include <thrust/reduce.h> // adicionado
#include <thrust/functional.h> // adicionado

using namespace std::chrono;

// Functor para calcular o quadrado (transform neste caso é elevação dos elementos do vetor ao quadrado)
struct square {
    __host__ __device__ // CPU e GPU
    float operator()(float x) const {
        return x * x;
    }
};

// feito para imprimir o tempo de execução de uma parte do código
void reportTime(const char* msg, steady_clock::duration span) {
    auto ms = duration_cast<milliseconds>(span);
    std::cout << msg << " - levou - " <<
    ms.count() << " milisegundos" << std::endl;
}

/* // IMPLEMENTE O CALCULO DA MAGNITUDE COM THRUST
float magnitude(thrust::device_vector<float>& v_d) {
    float result;

    // faz a transformação square
    thrust::transform(v_d.begin(), v_d.end(), v_d.begin(), square()); // aplica a transformação da função square para elevar os elementos ao quadrado

    // faz a redução para obter a magnitude
    result = std::sqrt(thrust::reduce(v_d.begin(), v_d.end(), 0.0f, thrust::plus<float>())); // soma dos elementos do vetor usando a função plus


    return result;
} */

// IMPLEMENTE O CALCULO DA MAGNITUDE COM THRUST E FUSION PARA OTIMIZAR TRANSFORM E REDUCE
float magnitude(thrust::device_vector<float>& v_d) {
    float result;

    // faz a redução para obter a magnitude
    result = std::sqrt(thrust::transform_reduce(v_d.begin(), v_d.end(), square(), 0.0f, thrust::plus<float>())); // soma dos elementos do vetor usando a função plus

    return result;
}


int main(int argc, char** argv) {
    if (argc != 2) {
        std::cerr << argv[0] << ": numero invalido de argumentos\n"; 
        std::cerr << "uso: " << argv[0] << "  tamanho do vetor\n"; 
        return 1;
    }
    int n = std::atoi(argv[1]); //numero de elementos
    steady_clock::time_point ts, te;

    // Cria um vetor em thrust na memória da GPU com n elementos do tipo float
    thrust::device_vector<float> v_d(n); 

    // Iniciliza o vetor
    ts = steady_clock::now();

    std::generate(v_d.begin(), v_d.end(), std::rand); // usado para preencher o vetor com valores aleatórios

    te = steady_clock::now();
    reportTime("Inicializacao", te - ts);

    // Aplica a transformação square
    ts = steady_clock::now();

    float len = magnitude(v_d);

    te = steady_clock::now();
    reportTime("Tempo para calculo", te - ts);

    std::cout << std::fixed << std::setprecision(4);
    std::cout << "Magnitude : " << sqrt(len) << std::endl;
}

// ---- cmd ----
// nvcc -arch=sm_70 -std=c++14 magnitude.cu -o magnitude
// ./magnitude